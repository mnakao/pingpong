#include <mpi.h>
#include <stdio.h>
extern "C" {
#include "common.h"
}

char send_buf[MAX_SIZE], recv_buf[MAX_SIZE];

int main(int argc, char** argv)
{
  int i, me, target;
  unsigned int size;
  double t;
  MPI_Status status;

  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &me);

  target = 1 - me;
  init_buf(send_buf, me);
  init_buf(recv_buf, target);
  
  char *send_buf_d, *recv_buf_d;
  hipMalloc((void **)&send_buf_d, MAX_SIZE);
  hipMalloc((void **)&recv_buf_d, MAX_SIZE);

  // Initialize
  hipMemcpy(send_buf_d, send_buf, MAX_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(recv_buf_d, recv_buf, MAX_SIZE, hipMemcpyHostToDevice);

  if(me==0) print_items();

  for(size=1;size<MAX_SIZE+1;size*=2){
    MPI_Barrier(MPI_COMM_WORLD);
    for(i=0;i<LOOP+WARMUP;i++){
      if(WARMUP == i)
        t = wtime();

      if(me == 0){
        MPI_Send(send_buf_d, size, MPI_CHAR, target, 9, MPI_COMM_WORLD);
        MPI_Recv(recv_buf_d, size, MPI_CHAR, target, 5, MPI_COMM_WORLD, &status);
      }
      else {
        MPI_Recv(recv_buf_d, size, MPI_CHAR, target, 9, MPI_COMM_WORLD, &status);
        MPI_Send(send_buf_d, size, MPI_CHAR, target, 5, MPI_COMM_WORLD);
      }
    }

    MPI_Barrier(MPI_COMM_WORLD);
    t = wtime() - t;
    if(me == 0)
      print_results(size, t);
  }

  hipFree(send_buf_d);
  hipFree(recv_buf_d);

  MPI_Finalize();
  return 0;
}

